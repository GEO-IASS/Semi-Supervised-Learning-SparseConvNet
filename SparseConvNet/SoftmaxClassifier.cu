#include "hip/hip_runtime.h"
#include "SoftmaxClassifier.h"
#include <iostream>
#include <vector>
#include <cassert>
#include "utilities.h"
#include <cfloat>

__global__ void dDerivativeOfCostWRTpreSoftmaxTopLevelWeights(
    int batchSize, float *topDelta, float *topGrid, int *labels, int N) {
  for (int k = batchSize/2; k < batchSize; k++) {    
    for (int i = threadIdx.x; i < N; i += NTHREADS) {      
      topDelta[k * N + i] = topGrid[k * N + i] - (i == labels[k]);
    }
  }
  for (int k = 0; k < batchSize/2; k++) {    
    for (int i = threadIdx.x; i < N; i += NTHREADS) {      
      topDelta[k * N + i] = 0;
    }
  }  
}

__global__ void mx_backprop(int batchSize, float* out_data, float* in_data,
    int N, float coe) {
  for (int index = threadIdx.x; index < batchSize*N/2; index += NTHREADS) {      
    
    int ind = (index / N) * N;
    float p = 1;
    for (int i = 0; i<N; i++){
	if ((ind+i) != index ){
	    p = p * (1 - in_data[ ind + i ]);
	}
    }

    float t1 = p;
    
    float p0;
    float t2 = 0;    
    for (int i = 0; i<N; i++){
	if ((ind+i) != index ){
	    p0 = 1;
	    for (int j = 0; j<N; j++){
		if ( (ind+j) != index && j != i ){
		    p0 = p0 * (1 - in_data[ ind + j ]);
		}
	    }
	  
	    t2 = t2 + p0 * in_data[ ind + i ];	  
	}
    }
    t2 = -t2;
    float v = (t1 + t2);  
        
    out_data[index] = -coe*v;    
    
  }
  for (int index = batchSize*N/2+threadIdx.x; index < batchSize*N; index += NTHREADS) {      
    out_data[index] = 0;	
  }
}

__global__ void ts_forward(int batchSize, float* in_data,
    int nt, int N, float* out_data) {  
  for (int index = threadIdx.x; index < batchSize*N*nt/2; index += NTHREADS) {          
    int ind1 = (nt*N)*(index /(nt*nt*N)) + index % (nt*N);
    int ind2 = N*(index /(N*nt)) + index % N;    
    out_data[index] = in_data[ind1] - in_data[ind2];
  }
  for (int index = batchSize*N*nt/2+threadIdx.x; index < batchSize*N*nt; index += NTHREADS) {          
    out_data[index] = 0;
  }
}

__global__ void ts_backprop(int batchSize, float* in_data,
    int nt, int N, float* out_data, float coe) {  
  for (int index = threadIdx.x; index < batchSize*N/2; index += NTHREADS) {     
    int ind1 = (nt*N)*(index /N) + index % N;    
    for (int i=0; i<nt; i++){
      out_data[index] = out_data[index] - coe*in_data[ind1 + N*i];       
    }
  }
}

__global__ void Softmaxb(int batchSize, float* dE_dx_l, float* y_l, float* dE_dy_l,
    int N) {
  for (int index = threadIdx.x; index < batchSize*N/2; index += NTHREADS) {  
    int tx = index % N;
    int ty = N * (index / N);
    
    float v = 0;
    for (int j = 0; j < N; j++) {
	v += dE_dy_l[j + ty] * ((j == tx) - y_l[j + ty]);
    }
    v *= y_l[index];        
    dE_dx_l[index] = dE_dx_l[index] + v;     
  } 
}

void SoftmaxClassifier(SpatiallySparseBatchInterface &input,
                       SpatiallySparseBatch &batch, int nTop,
                       cudaMemStream &memStream, Params *prms) {
  // Assume no dropout in the output layer! nClasses:=input.nFeatures.
  assert(batch.batchSize == input.nSpatialSites);
  assert(input.nFeatures == input.featuresPresent.size());  
        
  int nt = prms->nt;  
  float lambda_mx = prms->lambda_mx;
  float lambda_ts = prms->lambda_ts;  
  
  vectorCUDA<float> tmp1;
  tmp1.resize(input.nSpatialSites * input.featuresPresent.size());        
  
  if (batch.type ==
      TRAINBATCH) { // Begin backprop. Top layer: d Cost / d SoftmaxInput    
    input.sub->dfeatures.resize(input.nSpatialSites * input.featuresPresent.size());
	
    dDerivativeOfCostWRTpreSoftmaxTopLevelWeights
            << <1, NTHREADS, 0, memStream.stream>>>
        (batch.batchSize, input.sub->dfeatures.dPtr(),
         input.sub->features.dPtr(), batch.labels.dPtr(), input.nFeatures);
    
    mx_backprop<< <1, NTHREADS, 0, memStream.stream>>>
        (batch.batchSize, tmp1.dPtr(),
         input.sub->features.dPtr(), input.nFeatures, lambda_mx);
    
  }  
  
  vectorCUDA<float> tmp2;
  tmp2.resize(input.nSpatialSites * nt * input.featuresPresent.size());
  
  
  ts_forward<< <1, NTHREADS, 0, memStream.stream>>>
      (batch.batchSize, input.sub->features.dPtr(), nt, input.nFeatures, tmp2.dPtr());    
  
  if (batch.type == TRAINBATCH){

    ts_backprop<< <1, NTHREADS, 0, memStream.stream>>>
        (batch.batchSize, tmp2.dPtr(), nt, input.nFeatures, tmp1.dPtr(), lambda_ts ); 
	    
    Softmaxb<< <1, NTHREADS, 0, memStream.stream>>>
        (batch.batchSize, input.sub->dfeatures.dPtr(),
         input.sub->features.dPtr(), tmp1.dPtr(), input.nFeatures);
	
  }  
  
  input.sub->features.copyToCPUAsync(memStream);
  batch.labels.copyToCPUAsync(memStream);
  tmp2.copyToCPUAsync(memStream);  
    
    
  for (int i=0; i < batch.batchSize*input.nFeatures; i++){    
    if (isnan(input.sub->features.hVector()[i])){
      std::cout << "Found NaN\n";
      exit(EXIT_FAILURE);
    }
  }
 
  
  float *probs = &input.sub->features.hVector()[0];
  for (int i = 0; i < batch.batchSize; ++i)
    batch.probabilities.push_back(std::vector<float>(
        probs + i * input.nFeatures, probs + (i + 1) * input.nFeatures));
  for (int i = 0; i < batch.batchSize; i++)
    batch.predictions.push_back(vectorTopIndices(batch.probabilities[i], nTop));

  if (batch.type != UNLABELEDBATCH) {
    batch.mistakes += batch.batchSize;
    for (int i = 0; i < batch.batchSize; i++) {
      /* batch.negativeLogLikelihood -=
          log(max(batch.probabilities[i][batch.labels.hVector()[i]], 1.0e-15)); */
      for (int j = 0; j < nTop; j++) {
        if (batch.predictions[i][j] == batch.labels.hVector()[i]) {
          batch.mistakes--;
        }
      }
    }
  }  
  
  if (batch.type == TRAINBATCH){
    float cost_ts = 0;  
    for (int i = 0; i < input.nSpatialSites * nt * input.featuresPresent.size()/2; i++){
      cost_ts = cost_ts + tmp2.hVector()[i]*tmp2.hVector()[i];
    }
    
    float cost_mx = 0;
    float p;
    for (int i = 0; i < batch.batchSize/2; i++){ 
      for (int j = 0; j < input.nFeatures; j++){
	p = 1;
	for (int k = 0; k < input.nFeatures; k++){
	  if (k != j){
	    p = p * (1 - batch.probabilities[i][k]);      
	  }
	}
	p = p * batch.probabilities[i][j];
	cost_mx = cost_mx + p;
      }
    }
    
    batch.negativeLogLikelihood += cost_ts;
  }
  
  input.sub->features.copyToGPUAsync(memStream);
  cudaCheckError();
}
